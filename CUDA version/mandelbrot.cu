
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 256
#define MaxColor 255

int *a;

__device__ double CxMin = -2.5;
__device__ double CxMax = 1.5;
__device__ double CyMin = -2.0;
__device__ double CyMax = 2.0;

__device__ int seriesConverges(int x, int y,int width){
	double Cx,Cy,PixelHeight,PixelWidth;
	double Zx,Zy,Zx2,Zy2;
	PixelWidth = (CxMax-CxMin)/width;
	PixelHeight = (CyMax-CyMin)/width;
	Cy = CyMin +x*PixelHeight;
	if (fabs(Cy) < PixelHeight/2) {
		Cy = 0.0;
	}
	Cx = CxMin + y*PixelWidth;
	int color = 1;
	Zx = 0.0;
	Zy = 0.0;
	Zx2 = 0.0;
	Zy2 = 0.0;
	for (int i = 0; i < 512 && ((Zx2+Zy2)<4); i++) {
		Zx2 = Zx*Zx;
		Zy2 = Zy*Zy;
		Zy *= Zx;
		Zy += Zy+Cy;
		Zx = Zx2-Zy2+Cx;
		color++;
	}
	return color;
}

__global__ void mandelKernel(int *a, int width) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < width) {
		for (int j = 0; j < width; j++) {
			a[i*width+j] = seriesConverges(i,j,width);
		}
	}
}	

void mandelDevice(int *a, int tam) {
	float time;
	hipEvent_t start, stop;
	int *aD;
	int size = N*N*tam*tam*sizeof(int);
	int tam2 = N*N*tam*tam;
	int bsize =ceil((float)tam2/(float)1024);
	dim3 bloques(bsize);
	dim3 hilos (1024);
	hipSetDevice(0);
	hipMalloc(&aD, size);
	hipMemcpy(aD, a, size, hipMemcpyDefault);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	mandelKernel<<<bloques, hilos>>>(aD,N*tam);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("%3.1f \n", time);
	hipMemcpy(a, aD, size, hipMemcpyDefault);
	hipFree(aD);
}

int main() {
	FILE  *fp;
	static unsigned char color[3];
	int colorValue;
	for(int counter = 50; counter<=50; counter++){
		a = (int *)calloc(N*N*counter*counter,sizeof(int));
		mandelDevice(a, counter);
		fp = fopen("mandelbrotSet2.ppm","wb");
		fprintf(fp,"P6\n #\n %d\n %d\n %d\n",N*counter,N*counter,MaxColor);
		for (int i = 0; i < N*counter; i++) {
			for(int j = 0; j < N*counter; j++) {
				colorValue = a[i*N*counter+j];
				color[0] = (unsigned char)colorValue%MaxColor;
				color[1] = (unsigned char)colorValue%MaxColor;
				color[2] = (unsigned char)colorValue%MaxColor;
				fwrite(color,1,3,fp);
			}
		}
		fclose(fp);
		free(a);
	}
	return 0;
}
